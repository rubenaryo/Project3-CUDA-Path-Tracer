#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/gather.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "bsdf.h"

#include <stb_image.h>
#include <stb_image_write.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static Light* dev_lights = NULL;
static glm::vec3* dev_vertices = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec3* dev_tangents = NULL;
static glm::vec2* dev_uvs = NULL;
static glm::uvec3* dev_indices = NULL;
static BVHNode* dev_bvhNodes = NULL;

int pathBufferIdx = 0;
static PathSegment* dev_paths[2] = { NULL, NULL };
static ShadeableIntersection* dev_intersections[2] = { NULL, NULL };

static int* dev_sortIndices = NULL;
static MaterialSortKey* dev_sortKeys = NULL;  // Parallel array of flags to mark material type.
static hipTextureObject_t* dev_textureObjs = NULL;
static hipTextureObject_t* dev_envMapObjs = NULL; // Allowed to remain NULL if scene does not support env map

// TODO: static variables for device memory, any extra info you need, etc
// ...

// 1D block for path tracing
static const int BLOCK_SIZE_1D = 128;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

__host__ bool AllocDeviceTexture(HostTextureHandle& h, bool envMap)
{
    if (h.texObj)
    {
        return true; // Texture is already loaded. Return true
    }

    // Load on host side
    int channels;
    void* h_data = nullptr;
    
    if (envMap)
        h_data = stbi_loadf(h.filePath.c_str(), &h.width, &h.height, &channels, 4);
    else
        h_data = stbi_load(h.filePath.c_str(), &h.width, &h.height, &channels, 4);

    if (!h_data) {
        printf("Failed to load texture: %s\n", h.filePath.c_str());
        return false; // Texture failed to load.
    }

    hipChannelFormatDesc channelDesc = envMap ? hipCreateChannelDesc<float4>() : hipCreateChannelDesc<uchar4>();
    hipMallocArray(&h.cudaArr, &channelDesc, h.width, h.height);

    // Copy to CUDA
    size_t pitch = h.width * 4;
    pitch *= envMap ? sizeof(float) : sizeof(unsigned char);

    hipMemcpy2DToArray(h.cudaArr, 0, 0, h_data,
        pitch,
        pitch,
        h.height,
        hipMemcpyHostToDevice);

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = h.cudaArr;

    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = envMap ? hipAddressModeClamp : hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = envMap ? hipReadModeElementType : hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;
    texDesc.sRGB = h.sRGB && !envMap ? 1 : 0;

    // Hold tex obj handle on host
    hipCreateTextureObject(&h.texObj, &resDesc, &texDesc, nullptr);

    // Free host alloc
    stbi_image_free(h_data);
    return true;
}

void initDeviceTextures()
{
    std::vector<hipTextureObject_t> tempHostArr;
    tempHostArr.reserve(hst_scene->textures.size());
    for (HostTextureHandle& h : hst_scene->textures)
    {
        if (AllocDeviceTexture(h, false))
        {
            assert(h.texObj != 0);
            tempHostArr.push_back(h.texObj);
        }
    }

    hipMalloc(&dev_textureObjs, hst_scene->textures.size() * sizeof(hipTextureObject_t));
    hipMemcpy(dev_textureObjs, tempHostArr.data(), tempHostArr.size() * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);

    if (!hst_scene->envMapHandle.filePath.empty()) // Scene verifies that this exists
    {
        if (AllocDeviceTexture(hst_scene->envMapHandle, true))
        {
            assert(hst_scene->envMapHandle.texObj != 0);
            static const int NUM_SUPPORTED_ENV_MAPS = 1;

            hipMalloc(&dev_envMapObjs, NUM_SUPPORTED_ENV_MAPS * sizeof(hipTextureObject_t));
            hipMemcpy(dev_envMapObjs, &hst_scene->envMapHandle.texObj, NUM_SUPPORTED_ENV_MAPS * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
        }
    }

    checkCUDAError("initDeviceTextures");
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths[0], pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_paths[1], pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_lights, scene->lights.size() * sizeof(Light));
    hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Light), hipMemcpyHostToDevice);
    
    // Copy over master mesh data
    {
        hipMalloc(&dev_vertices, scene->masterMeshData.vertices.size() * sizeof(glm::vec3));
        hipMalloc(&dev_normals, scene->masterMeshData.normals.size() * sizeof(glm::vec3));
        hipMalloc(&dev_tangents, scene->masterMeshData.tangents.size() * sizeof(glm::vec3));
        hipMalloc(&dev_uvs, scene->masterMeshData.uvs.size() * sizeof(glm::vec2));
        hipMalloc(&dev_indices, scene->masterMeshData.indices.size() * sizeof(glm::uvec3));

        hipMemcpy(dev_vertices, scene->masterMeshData.vertices.data(), scene->masterMeshData.vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMemcpy(dev_normals, scene->masterMeshData.normals.data(), scene->masterMeshData.normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMemcpy(dev_tangents, scene->masterMeshData.tangents.data(), scene->masterMeshData.tangents.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMemcpy(dev_uvs, scene->masterMeshData.uvs.data(), scene->masterMeshData.uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
        hipMemcpy(dev_indices, scene->masterMeshData.indices.data(), scene->masterMeshData.indices.size() * sizeof(glm::uvec3), hipMemcpyHostToDevice);
    }

    hipMalloc(&dev_bvhNodes, scene->bvhNodes.size() * sizeof(BVHNode));
    hipMemcpy(dev_bvhNodes, scene->bvhNodes.data(), scene->bvhNodes.size() * sizeof(BVHNode), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections[0], pixelcount * sizeof(ShadeableIntersection));
    hipMalloc(&dev_intersections[1], pixelcount * sizeof(ShadeableIntersection));
    thrust::fill(thrust::device, dev_intersections[0], dev_intersections[0] + pixelcount, ShadeableIntersection());
    thrust::fill(thrust::device, dev_intersections[1], dev_intersections[1] + pixelcount, ShadeableIntersection());

    hipMalloc(&dev_sortKeys, pixelcount * sizeof(MaterialSortKey));
    thrust::fill(thrust::device, dev_sortKeys, dev_sortKeys + pixelcount, SORTKEY_INVALID);
    
    hipMalloc(&dev_sortIndices, pixelcount * sizeof(int));
    hipMemset(dev_sortIndices, 0, pixelcount * sizeof(int));

    initDeviceTextures();

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths[0]);
    hipFree(dev_paths[1]);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_lights);
    hipFree(dev_vertices);
    hipFree(dev_normals);
    hipFree(dev_tangents);
    hipFree(dev_uvs);
    hipFree(dev_indices);
    hipFree(dev_bvhNodes);
    hipFree(dev_intersections[0]);
    hipFree(dev_intersections[1]);
    hipFree(dev_sortKeys);
    hipFree(dev_sortIndices);
    hipFree(dev_textureObjs);
    hipFree(dev_envMapObjs);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= cam.resolution.x || y >= cam.resolution.y)
        return;

    int index = x + (y * cam.resolution.x);
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);
    segment.Lo = glm::vec3(0.0f);

#if STOCHASTIC_AA
    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
    thrust::uniform_real_distribution<float> uH(-0.5f, 0.5f);

    segment.ray.direction = glm::normalize(cam.view
        - cam.right * cam.pixelLength.x * ((float)x + uH(rng) - (float)cam.resolution.x * 0.5f)
        - cam.up * cam.pixelLength.y * ((float)y + uH(rng) - (float)cam.resolution.y * 0.5f)
    );
#else
    segment.ray.direction = glm::normalize(cam.view
        - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
        - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
    );
#endif

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
}

__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    const SceneData sceneData,
    ShadeableIntersection* intersections,
    hipTextureObject_t* envMaps // Maybe temp
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index >= num_paths)
        return;
    
    PathSegment& path = pathSegments[path_index];
    ShadeableIntersection& result = intersections[path_index];
    sceneIntersect(path, sceneData, result, envMaps);
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];

        const glm::vec3 RADIANCE_UPPER_BOUND(1000000000000.0f);
        assert(glm::all(glm::lessThan(iterationPath.Lo, RADIANCE_UPPER_BOUND)));
        image[iterationPath.pixelIndex] += glm::clamp(iterationPath.Lo, glm::vec3(0.0f), RADIANCE_UPPER_BOUND);
    }
}

//struct MaterialIdComp {
//    
//    using PathIsectTuple = thrust::tuple<PathSegment, ShadeableIntersection>;
//    
//    __host__ __device__
//    bool operator()(const PathIsectTuple& a, const PathIsectTuple& b) const {
//        return thrust::get<1>(a).materialId < thrust::get<1>(b).materialId;
//    }
//};
struct NonTerminated {
    __host__ __device__
        bool operator()(PathSegment& ps) {
        return ps.remainingBounces > 0;
    }
};

struct Terminated {
    __host__ __device__
        bool operator()(const PathSegment& ps) const {
        return ps.remainingBounces == 0;
    }
};

struct IsectSortKeyComp {
    __host__ __device__
        bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
        return a.matSortKey < b.matSortKey;
    }
};

__global__ void testKernel(int N, PathSegment* paths, ShadeableIntersection* isects, MaterialSortKey* sortKeys)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    MaterialSortKey thisMat = sortKeys[idx];

    const PathSegment path = paths[idx];
    paths[idx] = path;

    

    MaterialSortKey* dummy = sortKeys + N;
}

/// Sorts the PathSegment and Intersection arrays by material type (dev_sortKeys)
/// Returns the new number of paths after discarding the non-intersections at the end of the array
__host__ int sortByMaterialType(int num_paths)
{
    typedef thrust::zip_iterator<cuda::std::tuple<PathSegment*, ShadeableIntersection*>> ZipIterator;

#if SORT_BY_ZIP_ITERATOR
    // Sort both arrays together as a zip_iterator
    // This also sorts the keys (dev_sortKeys)
    // This is really slow!
    ZipIterator zip_it = thrust::make_zip_iterator(thrust::make_tuple(dev_paths[pathBufferIdx], dev_intersections[pathBufferIdx]));
    thrust::sort_by_key(thrust::device, dev_sortKeys, dev_sortKeys + num_paths, zip_it);
#else
    int src = pathBufferIdx;
    int dst = 1 - pathBufferIdx;

    // Indices used for isect sorting
    thrust::sequence(thrust::device, dev_sortIndices, dev_sortIndices + num_paths);

    // Sort the indices (representing path/isect elements) based on the previously gathered sortkeys
    thrust::sort_by_key(thrust::device, dev_sortKeys, dev_sortKeys + num_paths, dev_sortIndices);

    // 
    thrust::gather(thrust::device, dev_sortIndices, dev_sortIndices + num_paths, dev_paths[src], dev_paths[dst]);
    thrust::gather(thrust::device, dev_sortIndices, dev_sortIndices + num_paths, dev_intersections[src], dev_intersections[dst]);

    // Write back
    // TODO: Swapping pathBufferIdx should work, but doesnt...
    hipMemcpy(dev_paths[src], dev_paths[dst], sizeof(PathSegment) * num_paths, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_intersections[src], dev_intersections[dst], sizeof(ShadeableIntersection) * num_paths, hipMemcpyDeviceToDevice);

    // This should work instead of memcpy, but doesn't for some reason ...
    //pathBufferIdx = dst;

#endif // 0

    int numBlocks = utilityCore::divUp(num_paths, BLOCK_SIZE_1D);

    // Binary search to find the partition point after which all paths are invalid
    MaterialSortKey* firstInvalid_it = thrust::lower_bound(thrust::device,
        dev_sortKeys, dev_sortKeys + num_paths, SORTKEY_INVALID);

    // Return the new number of paths
    return firstInvalid_it - dev_sortKeys;
}


// Note: Assumes dev_sortKeys has been sorted already by sortByMaterialType
__host__ void shadeByMaterialType(int num_paths, int iter, int depth, const SceneData& sd)
{
    using utilityCore::divUp;

    // For every material type, launch its corresponding kernel

    // These args stay the same across all rays.
    ShadeKernelArgs skArgs;
    skArgs.iter = iter;
    skArgs.depth = depth;
    skArgs.materials = dev_materials;
    skArgs.textures = dev_textureObjs;
    skArgs.envMaps = dev_envMapObjs;
    skArgs.sceneData = sd;

    void* cudaKernelArgs[] = { &skArgs };

    dim3 numBlocks;
    int prev_end = 0;
    for (unsigned int m = MT_FIRST; m < MT_COUNT; ++m)
    {
        // Find the index range for this material
        MaterialSortKey maxKey = BuildSortKey((MaterialType)m, UINT16_MAX);
        int mt_end = thrust::upper_bound(thrust::device, dev_sortKeys + prev_end, dev_sortKeys + num_paths, maxKey) - dev_sortKeys;
        int mt_start = prev_end;
        int mt_count = mt_end - mt_start;
        
        // If there are rays for this material type, dispatch them all together in the same kernel
        if (mt_count)
        {
            skArgs.num_paths = mt_count;
            skArgs.pathSegments = dev_paths[pathBufferIdx] + mt_start;
            skArgs.shadeableIntersections = dev_intersections[pathBufferIdx] + mt_start;

            numBlocks.x = divUp(mt_count, BLOCK_SIZE_1D);
            ShadeKernel sk = getShadingKernelForMaterial((MaterialType)m);
            hipLaunchKernel(reinterpret_cast<const void*>(sk), numBlocks, BLOCK_SIZE_1D, cudaKernelArgs, 0, nullptr);
            checkCUDAError("hipLaunchKernel: shadingKernel");
        }

        prev_end = mt_end;
    }
}

__host__ void shadeLegacy(int num_paths, int iter, int depth)
{
    // TODO: compare between directly shading the path segments and shading
    // path segments that have been reshuffled to be contiguous in memory.

    ShadeKernelArgs skArgs = {
          iter
        , num_paths
        , depth
        , dev_intersections[pathBufferIdx]
        , dev_paths[pathBufferIdx]
        , dev_materials
        // TODO: Need to pass SceneData here
    };
    void* cudaKernelArgs[] = { &skArgs };

    dim3 numblocksPathSegmentTracing = (num_paths + BLOCK_SIZE_1D - 1) / BLOCK_SIZE_1D;
    ShadeKernel sk = getShadingKernelForMaterial(MT_DIFFUSE);
    hipLaunchKernel(reinterpret_cast<const void*>(sk), numblocksPathSegmentTracing, BLOCK_SIZE_1D, cudaKernelArgs, 0, nullptr);
    checkCUDAError("hipLaunchKernel: shadingKernel");
}

__host__ int cullTerminatedPaths(int num_paths)
{
    auto dev_path_end = thrust::partition(thrust::device, dev_paths[pathBufferIdx], dev_paths[pathBufferIdx] + num_paths, NonTerminated());
    //auto dev_path_end = thrust::remove_if(thrust::device, dev_paths, dev_paths + num_paths, Terminated());
    return dev_path_end - dev_paths[pathBufferIdx];
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int maxDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    SceneData sd;
    sd.geoms = dev_geoms;
    sd.geoms_size = hst_scene->geoms.size();
    sd.lights = dev_lights;
    sd.lights_size = hst_scene->lights.size();
    sd.vertices = dev_vertices;
    sd.vertices_size = hst_scene->masterMeshData.vertices.size();
    sd.normals = dev_normals;
    sd.normals_size = hst_scene->masterMeshData.normals.size();
    sd.tangents = dev_tangents;
    sd.tangents_size = hst_scene->masterMeshData.tangents.size();
    sd.uvs = dev_uvs;
    sd.uvs_size = hst_scene->masterMeshData.uvs.size();
    sd.indices = dev_indices;
    sd.indices_size = hst_scene->masterMeshData.indices.size();
    sd.bvhNodes = dev_bvhNodes;
    sd.bvhNodes_size = hst_scene->bvhNodes.size();

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, maxDepth, dev_paths[pathBufferIdx]);
    checkCUDAError("generate camera ray");

    int depth = 0;
    int num_paths = pixelcount;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        thrust::fill(thrust::device, dev_intersections[0], dev_intersections[0] + pixelcount, ShadeableIntersection());
        thrust::fill(thrust::device, dev_intersections[1], dev_intersections[1] + pixelcount, ShadeableIntersection());

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + BLOCK_SIZE_1D - 1) / BLOCK_SIZE_1D;
        computeIntersections<<<numblocksPathSegmentTracing, BLOCK_SIZE_1D>>> (
            depth,
            num_paths,
            dev_paths[pathBufferIdx],
            sd,
            dev_intersections[pathBufferIdx],
            dev_envMapObjs
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();

    #if STREAM_COMPACTION || MATERIAL_SORT
        // Flag intersections by material type. We will use this to sort the path and isect arrays
        generateSortKeys<<<numblocksPathSegmentTracing, BLOCK_SIZE_1D>>>(num_paths, dev_intersections[pathBufferIdx], dev_materials, dev_sortKeys);
        checkCUDAError("generateSortKeys");
    #endif

    #if MATERIAL_SORT
        int new_num_paths = sortByMaterialType(num_paths);
        checkCUDAError("sortByMaterialType");
        // TODO_WAVEFRONT: use new_num_paths to determine how many rays to regenerate.
        num_paths = new_num_paths;

        if (num_paths == 0)
            break;
    #endif

    #if MATERIAL_SORT
        shadeByMaterialType(num_paths, iter, depth, sd);
        checkCUDAError("shadeByMaterialType");
    #else
        shadeLegacy(num_paths, iter, depth);
    #endif

    #if STREAM_COMPACTION
        num_paths = cullTerminatedPaths(num_paths);
        // TODO_WAVEFRONT: Regenerate paths
    #endif
        iterationComplete = depth >= maxDepth || num_paths == 0;
        depth++;

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + BLOCK_SIZE_1D - 1) / BLOCK_SIZE_1D;
    finalGather<<<numBlocksPixels, BLOCK_SIZE_1D>>>(pixelcount, dev_image, dev_paths[pathBufferIdx]);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
