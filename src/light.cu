#include "hip/hip_runtime.h"
#include "light.h"

#include "bsdf.h"
#include <cmath>

__device__ glm::vec3 DirectSampleAreaLight(glm::vec3 view_point, glm::vec3 view_nor, const Light& chosenLight, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf, float& out_distToLight)
{
	GeomType gt = chosenLight.geomType;

	switch (gt)
	{
	case GT_RECT:
		{
			float scaleX = chosenLight.scale.x;
			float scaleZ = chosenLight.scale.z;
			thrust::uniform_real_distribution<float> u01(-0.5f, 0.5f);
			glm::vec4 randPosLocal(u01(rng), u01(rng), u01(rng), 0.0f);
			randPosLocal = glm::normalize(randPosLocal);
			randPosLocal.w = 1.0f;
			glm::vec4 norLocal(0.0f, 0.0f, 1.0f, 0.0f);

			glm::vec3 randPosWorld(chosenLight.transform * randPosLocal);
			glm::vec3 norWorld(chosenLight.transform * norLocal);

			float surfaceArea = (scaleX) * (scaleZ);
			float areaPDF = 1.0f / surfaceArea;

			glm::vec3 lightToSurface = view_point - randPosWorld;
			float r2 = glm::dot(lightToSurface, lightToSurface);
			out_distToLight = glm::sqrt(r2);
			
			//if (r2 < FLT_EPSILON)
			//{
			//	out_pdf = 0.0f;
			//	out_wiW = glm::normalize(-lightToSurface);
			//	return glm::vec3(0.0f);
			//}

			lightToSurface *= glm::inversesqrt(r2); // normalize
			out_wiW = -lightToSurface;
			float cosTheta = glm::abs(glm::dot(norWorld, lightToSurface));

			out_pdf = (r2 / cosTheta) * areaPDF;

			return cosTheta * chosenLight.emittance * chosenLight.color;

		}
		break;
	default:
		// TODO
	}

	float Le = chosenLight.emittance;
	return Le * numLights * chosenLight.color;
}

__device__ glm::vec3 Sample_Li(glm::vec3 view_point, glm::vec3 nor, Light* lights, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf, float& out_distToLight)
{
	thrust::uniform_int_distribution<int> iu0N(0, numLights-1);
	int randomLightIndex = iu0N(rng);
	const Light chosenLight = lights[randomLightIndex];

	return DirectSampleAreaLight(view_point, nor, chosenLight, numLights, rng, out_wiW, out_pdf, out_distToLight);
}
