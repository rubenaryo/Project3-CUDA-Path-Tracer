#include "hip/hip_runtime.h"
#include "light.h"

//#include "bsdf.h"
#include "intersections.h"
#include <cmath>

__device__ float PowerHeuristic(int nf, float fPdf, int ng, float gPdf) {
    float f = nf * fPdf;
    float g = ng * gPdf;
    return (f * f) / (f * f + g * g);
}

__device__ bool areaLightIntersect(const Light& chosenLight, Ray r, ShadeableIntersection& out_isect)
{
    out_isect.t = FLT_MAX;
    switch (chosenLight.geomType)
    {
    case GT_RECT:
    {
        glm::vec3 pos(0.0f);
        glm::vec3 nor(0.0f, 0.0f, 1.0f);
        glm::vec2 halfSideLengths = glm::vec2(chosenLight.scale.x, chosenLight.scale.z); // TODO: Maybe get this from the light's scale?
        glm::vec3 toLightLocal;
        glm::vec2 uv;
        float d = rectIntersectionTest(pos, nor,
            halfSideLengths.x, halfSideLengths.y,
            r, chosenLight.inverseTransform, toLightLocal, uv);

        if (d > (FLT_MAX - FLT_EPSILON))
            return false;

        glm::vec3 toLightWorld = multiplyMV(chosenLight.inverseTransform, glm::vec4(toLightLocal, 1.0f));

        out_isect.t = glm::length(toLightWorld); // TODO: This feels wrong, isn't d in local space?
        out_isect.surfaceNormal = glm::vec3(glm::normalize(chosenLight.invTranspose * glm::vec4(0.0f, 0.0f, 1.0f, 0.0f)));
        // out_isect.Le = light.Le;
        // out_isect.obj_ID = light.ID;
    }
        break;
    default:
        // Unsupported
        break;
    }

    return out_isect.t < (FLT_MAX - FLT_EPSILON);
}

//__device__ float Pdf_Rect(const glm::vec3& halfSideLengths, const glm::vec3& view_point, const glm::vec3& light_point)

__device__ float Pdf_Rect(const Light& chosenLight, const glm::vec3& view_point, const glm::vec3& light_point, const glm::vec3& norW)
{
    using namespace glm;

    float scaleX = chosenLight.scale.x;
    float scaleZ = chosenLight.scale.z;

    float surfaceArea = (scaleX) * (scaleZ);
    float areaPDF = 1.0 / surfaceArea;

    vec3 lightToSurface = view_point - light_point;
    vec3 normalizedLightToSurface = normalize(lightToSurface);
    //vec3 norWorld = (chosenLight.transform * vec4(nor, 0.0)).xyz;
    float cosTheta = abs(dot(normalize(norW), normalizedLightToSurface));
    float r = length(lightToSurface);

    //if (cosTheta < 0.01)
    //    return 0.0;

    return (r * r / cosTheta) * areaPDF;
}

__device__ float Pdf_Li(const Light& chosenLight, const glm::vec3& view_point, const glm::vec3& norW, const glm::vec3& wiW)
{
    //Ray ray = SpawnRay(view_point, wiW);
    Ray ray;
    ray.direction = wiW;
    ray.origin = view_point + wiW * 0.001f;

    ShadeableIntersection isect;
    if (!areaLightIntersect(chosenLight, ray, isect))
        return 0.0f; // Didn't hit anything.

    glm::vec3 light_point = ray.origin + isect.t * wiW;

    switch (chosenLight.geomType)
    {
    case GT_RECT:
        return Pdf_Rect(chosenLight, view_point, light_point, norW);
        break;
    }

    return 0.0f;
}

__device__ glm::vec3 DirectSampleAreaLight(glm::vec3 view_point, glm::vec3 view_nor, const Light& chosenLight, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf, float& out_distToLight)
{
    GeomType gt = chosenLight.geomType;

    switch (gt)
    {
    case GT_RECT:
        {
            thrust::uniform_real_distribution<float> uH(-0.5f, 0.5f);
            glm::vec4 randPosLocal(uH(rng), uH(rng), 0.0f, 1.0f);
            glm::vec4 norLocal(0.0f, 0.0f, 1.0f, 0.0f);

            glm::vec3 randPosWorld(chosenLight.transform * randPosLocal);
            glm::vec3 norWorld(chosenLight.transform * norLocal);

            glm::vec3 edge1 = glm::vec3(chosenLight.transform * glm::vec4(1.0f, 0.0f, 0.0f, 0.0f));
            glm::vec3 edge2 = glm::vec3(chosenLight.transform * glm::vec4(0.0f, 1.0f, 0.0f, 0.0f));

            // Area is magnitude of cross product
            float surfaceArea = glm::length(glm::cross(edge1, edge2));
            if (surfaceArea < FLT_MAX)
            {
                // surface area is too small, early out.
                out_pdf = 0.0f;
                return glm::vec3(0.0f);
            }

            glm::vec3 toLightW = view_point - randPosWorld;
            float distToLightSq = glm::dot(toLightW, toLightW);
            float disttoLight = glm::sqrt(distToLightSq);
            
            if (distToLightSq < FLT_EPSILON)
            {
            	out_pdf = 0.0f;
            	out_wiW = glm::normalize(-toLightW);
            	return glm::vec3(0.0f);
            }

            toLightW *= glm::inversesqrt(distToLightSq); // normalize
            float cosTheta = (glm::dot(norWorld, toLightW));
            if (cosTheta < 0.0f)
            {
                out_pdf = 0.0f;
                return glm::vec3(0.0f);
            }
            
            out_distToLight = disttoLight;
            out_wiW = toLightW;
            out_pdf = distToLightSq / (cosTheta * surfaceArea);

            return numLights * chosenLight.emittance * chosenLight.color;
        }
        break;
    default:
        // TODO
    }

    float Le = chosenLight.emittance;
    return Le * numLights * chosenLight.color;
}

__device__ glm::vec3 Sample_Li(glm::vec3 view_point, glm::vec3 nor, const Light& chosenLight, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf, float& out_distToLight)
{
    return DirectSampleAreaLight(view_point, nor, chosenLight, numLights, rng, out_wiW, out_pdf, out_distToLight);
}
