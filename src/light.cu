#include "hip/hip_runtime.h"
#include "light.h"

#include "bsdf.h"
#include <cmath>

__device__ glm::vec3 DirectSampleAreaLight(glm::vec3 view_point, glm::vec3 view_nor, const Light& chosenLight, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf)
{
	GeomType gt = chosenLight.geomType;

	switch (gt)
	{
	case GT_RECT:
		{
			float scaleX = chosenLight.scale.x;
			float scaleZ = chosenLight.scale.z;
			thrust::uniform_real_distribution<float> u01(-0.5f, 0.5f);
			glm::vec4 randPosLocal(u01(rng), u01(rng), u01(rng), 0.0f);
			randPosLocal = glm::normalize(randPosLocal);
			glm::vec4 norLocal(0.0f, 0.0f, 1.0f, 0.0f);

			glm::vec3 randPosWorld(chosenLight.transform * randPosLocal);
			glm::vec3 norWorld(chosenLight.transform * norLocal);

			float surfaceArea = (scaleX) * (scaleZ);
			float areaPDF = 1.0f / surfaceArea;

			glm::vec3 lightToSurface = view_point - randPosWorld;
			float r2 = glm::dot(lightToSurface, lightToSurface);
			
			//if (r2 < FLT_EPSILON)
			//{
			//	out_pdf = 0.0f;
			//	out_wiW = glm::normalize(-lightToSurface);
			//	return glm::vec3(0.0f);
			//}

			lightToSurface *= glm::inversesqrt(r2); // normalize
			out_wiW = -lightToSurface;
			float cosTheta = glm::abs(glm::dot(norWorld, lightToSurface));

			//if (cosTheta < FLT_EPSILON)
			//	out_pdf = 0.0f;
			//else
				out_pdf = (r2 / cosTheta) * areaPDF;

			return chosenLight.emittance * cosTheta * chosenLight.color;

		}
		break;
	default:
		// TODO
	}

	float Le = chosenLight.emittance;
	return Le * numLights * chosenLight.color;
}

__device__ glm::vec3 Sample_Li(glm::vec3 view_point, glm::vec3 nor, Light* lights, int numLights, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf)
{
	thrust::uniform_int_distribution<int> iu0N(0, numLights-1);
	int randomLightIndex = iu0N(rng);
	const Light chosenLight = lights[randomLightIndex];

	return DirectSampleAreaLight(view_point, nor, chosenLight, numLights, rng, out_wiW, out_pdf);
}
