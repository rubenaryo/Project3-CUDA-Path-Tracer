#include "hip/hip_runtime.h"
#include "intersections.h"

/// Kernel to label each intersection with additional information to be used for ray sorting and discarding
__global__ void generateSortKeys(int N, const ShadeableIntersection* isects, Material* mats, MaterialSortKey* sortKeys)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    const ShadeableIntersection isect = isects[idx];
    if (isect.t > FLT_EPSILON)
    {
        const Material mat = mats[isect.materialId];
        sortKeys[idx] = BuildSortKey(mat.type, isect.materialId);
    }
    else
    {
        sortKeys[idx] = SORTKEY_INVALID;
    }
}

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = glm::min(t1, t2);
        outside = true;
    }
    else
    {
        t = glm::max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__host__ __device__ float triangleIntersectionTest(Geom tri, Ray r, glm::vec3& intersectionPoint, glm::vec3& normal, bool& outside)
{
    //bool result = glm::intersectRayTriangle(r.origin, r.direction, )
    return 0;
}


// From CIS 561
__host__ __device__ bool intersectRayTriangle_MollerTrumbore(const Ray& ray,
    const glm::vec3& v0, const glm::vec3& v1, const glm::vec3& v2,
    float& t, glm::vec3& isect, glm::vec3& nor, glm::vec2& uv)
{
    glm::vec3 edge1 = v1 - v0;
    glm::vec3 edge2 = v2 - v0;
    glm::vec3 h = glm::cross(ray.direction, edge2);
    float a = glm::dot(edge1, h);

    if (a > -FLT_EPSILON && a < FLT_EPSILON) 
        return false;

    float f = 1.0f / a;
    glm::vec3 s = ray.origin - v0;
    float u = f * glm::dot(s, h);

    if (u < 0.0f || u > 1.0f)
        return false;

    glm::vec3 q = glm::cross(s, edge1);
    float v = f * glm::dot(ray.direction, q);

    if (v < 0.0f || u + v > 1.0f)
        return false;

    t = f * glm::dot(edge2, q);

    if (t > FLT_EPSILON)
    {
        isect = ray.origin + ray.direction * t;
        uv = glm::vec2(u, v);
        nor = glm::normalize(glm::cross(edge1, edge2));
        return true;
    }

    return false;
}

__host__ __device__  float meshIntersectionTest(Geom meshGeom, const Mesh mesh, Ray r, glm::vec3& intersectionPoint, glm::vec3& normal, glm::vec2& uv, bool& outside)
{
    glm::vec3 ro = multiplyMV(meshGeom.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(meshGeom.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray testRay;
    testRay.origin = ro;
    testRay.direction = rd;

    outside = true;
    bool hitAnything = false;
    float tmin = FLT_MAX;
    int closestIdx = -1;

    glm::vec3 closestPos;
    glm::vec3 closestNor;
    glm::vec2 closestUV;

    for (uint32_t i = 0; (i+2) < mesh.vtx_count; i += 3)
    {
        glm::vec3 v0 = mesh.vtx[i];
        glm::vec3 v1 = mesh.vtx[i + 1];
        glm::vec3 v2 = mesh.vtx[i + 2];

        float t;
        glm::vec3 isectPos;
        glm::vec3 isectNor;
        glm::vec2 isectUV;
        //bool hit = glm::intersectRayTriangle<glm::vec3>(r.origin, r.direction, v0, v1, v2, baryPosition);
        bool hit = intersectRayTriangle_MollerTrumbore(testRay, v0, v1, v2, t, isectPos, isectNor, isectUV);
        if (hit && t < tmin)
        {
            hitAnything = true;
            tmin = t;
            closestIdx = i;
            closestPos = isectPos;
            closestNor = isectNor;
            closestUV  = isectUV;
        }
    }

    if (hitAnything)
    {
        intersectionPoint = glm::vec3(meshGeom.transform * glm::vec4(closestPos, 1.0f));
        normal = glm::vec3(meshGeom.invTranspose * glm::vec4(closestNor, 0.0f));
        normal = glm::normalize(normal);

        uv = glm::vec2(0.f);
        if (0 < closestIdx && closestIdx < (mesh.uvs_count - 2))
        {
            float u = closestUV.x;
            float v = closestUV.y;
            float w = 1.0f - u - v;

            // Barycentric interp to get uv coords
            uv = w * mesh.uvs[closestIdx] +
                 u * mesh.uvs[closestIdx + 1] +
                 v * mesh.uvs[closestIdx + 2];
        }

        return glm::length(r.origin - intersectionPoint);
    }

    return -1.0f;
}

__device__ void sceneIntersect(PathSegment& path, const Geom* geoms, int geoms_size, const Mesh* meshes, int meshes_size, ShadeableIntersection& result)
{
    float t;
    glm::vec3 intersect_point;
    glm::vec3 normal;
    float t_min = FLT_MAX;
    int hit_geom_index = -1;
    MaterialID hitMaterial = -1;
    bool outside = true;

    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;

    const PathSegment pathCopy = path;

    // naive parse through global geoms

    for (int i = 0; i < geoms_size; i++)
    {
        const Geom geom = geoms[i];

        if (geom.type == GT_CUBE)
        {
            t = boxIntersectionTest(geom, pathCopy.ray, tmp_intersect, tmp_normal, outside);
        }
        else if (geom.type == GT_SPHERE)
        {
            t = sphereIntersectionTest(geom, pathCopy.ray, tmp_intersect, tmp_normal, outside);
        }
        else if (geom.type == GT_MESH)
        {
            const Mesh mesh = meshes[geom.meshId];
            glm::vec2 uv;
            t = meshIntersectionTest(geom, mesh, pathCopy.ray, tmp_intersect, tmp_normal, uv, outside);

            
        }
        // TODO: add more intersection tests here... triangle? metaball? CSG?

        // Compute the minimum t from the intersection tests to determine what
        // scene geometry object was hit first.
        if (t > 0.0f && t_min > t)
        {
            t_min = t;
            hit_geom_index = i;
            hitMaterial = geom.materialid;
            intersect_point = tmp_intersect;
            normal = tmp_normal;
        }
    }

    if (hit_geom_index == -1)
    {
        result.t = -1.0f;
        path.color = glm::vec3(0.0f); // This gmem read might be really bad.
    }
    else
    {
        // The ray hits something
        result.t = t_min;
        result.materialId = hitMaterial;
        result.surfaceNormal = normal;
    }
}

__device__ void lightsIntersect(PathSegment& path, const Light* lights, int lights_size, ShadeableIntersection& result, LightID& resultId)
{
    // TODO_MIS
}
