#include "hip/hip_runtime.h"
#include "bsdf.h"

#include "light.h"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

__device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

inline __device__ void coordinateSystem(const glm::vec3& v1, glm::vec3& v2, glm::vec3& v3)
{
    if (glm::abs(v1.x) > glm::abs(v1.y))
        v2 = glm::vec3(-v1.z, 0, v1.x) / glm::sqrt(v1.x * v1.x + v1.z * v1.z);
    else
        v2 = glm::vec3(0, v1.z, -v1.y) / glm::sqrt(v1.y * v1.y + v1.z * v1.z);
    v3 = glm::cross(v1, v2);
}

inline __device__ glm::mat3 LocalToWorld(glm::vec3 nor)
{
    glm::vec3 tan, bit;
    coordinateSystem(nor, tan, bit);
    return glm::mat3(tan, bit, nor);
}


inline __device__ glm::mat3 WorldToLocal(glm::vec3 nor) {
    return glm::transpose(LocalToWorld(nor));
}

inline __device__ Ray SpawnRay(const glm::vec3& pos, const glm::vec3& wi)
{
    Ray r;
    r.origin = pos + wi * 0.001f;
    r.direction = wi;
    return r;
}

////////////////////////
// PDF functions

__device__ float squareToHemisphereCosinePDF(const glm::vec3& sampleL)
{
    return sampleL.z * INV_PI;
}

__device__ float Pdf(MaterialType matType, glm::vec3 norW, glm::vec3 woW, glm::vec3 wiW)
{
    glm::vec3 nor = norW;
    glm::mat3 world2Local = WorldToLocal(nor);
    glm::vec3 wo = world2Local * woW;
    glm::vec3 wi = world2Local * wiW;

    switch (matType)
    {
    case MT_DIFFUSE:
        return squareToHemisphereCosinePDF(wi);
    case MT_SPECULAR:
    case MT_REFRACTIVE:
        return 0.0f; // Spec goes directly to the light, and refractive goes into the material.
    }

    return 0.0f; // Unhandled material.
}

////////////////////////
// f / Samplef functions

inline __device__ glm::vec3 f_diffuse(const glm::vec3& albedo)
{
    return albedo * INV_PI;
}

inline __device__ glm::vec3 f_spec(const glm::vec3& albedo, const glm::vec3& wiW, const glm::vec3& norW)
{
    float absCosTheta = glm::abs(glm::dot(wiW, norW));
    if (absCosTheta < FLT_EPSILON)
        return glm::vec3(0.0f);

    return albedo / absCosTheta;
}

inline __device__ glm::vec3 Sample_f_diffuse(const glm::vec3& albedo, const glm::vec3& norW, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf)
{
    out_wiW = calculateRandomDirectionInHemisphere(norW, rng);
    out_pdf = glm::abs(glm::dot(out_wiW, norW)) * INV_PI;
    return f_diffuse(albedo);
}

inline __device__ glm::vec3 Sample_f_specular(const glm::vec3& albedo, const glm::vec3& woW, const glm::vec3& norW, glm::vec3& out_wiW, float& out_pdf)
{
    out_wiW = glm::reflect(woW, norW);
    out_pdf = 1.0f;
    return f_spec(albedo, out_wiW, norW);
}

__device__ bool SolveDirectLighting(const SceneData& sd, ShadeableIntersection isect, glm::vec3 view_point, thrust::default_random_engine& rng, glm::vec3& out_radiance, glm::vec3& out_wiW, float& out_pdf)
{
    int numLights = sd.lights_size;
    thrust::uniform_int_distribution<int> iu0N(0, numLights - 1);
    int randomLightIndex = iu0N(rng);
    const Light chosenLight = sd.lights[randomLightIndex];
    glm::vec3 norW = isect.surfaceNormal;

    float pdf_Li;
    float distToLight;
    glm::vec3 wiW_Li;

    glm::vec3 liResult = Sample_Li(view_point, norW, chosenLight, numLights, rng, wiW_Li, pdf_Li, distToLight);
    if (pdf_Li < FLT_EPSILON)
        return false;

    // Test occlusion
    PathSegment shadowPath;
    shadowPath.ray = SpawnRay(view_point, wiW_Li);
    ShadeableIntersection shadowTestResult;
    sceneIntersect(shadowPath, sd, shadowTestResult, chosenLight.geomId);
    
    if (shadowTestResult.t >= 0.0f && shadowTestResult.t < (distToLight - FLT_EPSILON))
        return false;

    out_radiance = liResult;
    out_wiW = wiW_Li;
    out_pdf = pdf_Li;

    return true;
}

/////////

#if STREAM_COMPACTION
#define HANDLE_MISS(idx, intersection, pathSegments) \
        assert((intersection).t > FLT_EPSILON);
#else
#define HANDLE_MISS(idx, intersection, pathSegments)              \
        if ((intersection).t <= 0.0f) {                               \
            (pathSegments)[(idx)].color = glm::vec3(0.0f);            \
            (pathSegments)[(idx)].remainingBounces = 0;               \
            return;                                                   \
        }
#endif

__global__ void skDiffuse(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);
    glm::vec3 view_point = path.ray.origin + intersection.t * path.ray.direction;
    glm::vec3 thisBounceRadiance(0.0f); // Comes from direct lighting only
    
    glm::vec3 wiW_bsdf;
    float pdf_bsdf;
    glm::vec3 bsdf;
    
    // BSDF Sampling
    bsdf = Sample_f_diffuse(material.color, intersection.surfaceNormal, rng, wiW_bsdf, pdf_bsdf);
    if (pdf_bsdf < FLT_EPSILON)
    {
        // Something went wrong, terminate
        args.pathSegments[idx].remainingBounces = 0;
        return;
    }

    glm::vec3 directRadiance;
    glm::vec3 wiW_Li;
    float pdf_Li;

    float lambert = glm::abs(glm::dot(intersection.surfaceNormal, wiW_bsdf));
    args.pathSegments[idx].throughput *= (bsdf/pdf_bsdf) * lambert;
    args.pathSegments[idx].prevBounceSample.pdf = pdf_bsdf;
    args.pathSegments[idx].prevBounceSample.matType = MT_DIFFUSE;
    args.pathSegments[idx].ray = SpawnRay(view_point, wiW_bsdf);
    args.pathSegments[idx].remainingBounces--;

    // Direct Light Sampling
    // Key difference using MIS: Accumulate direct lighting radiance here.
    glm::vec3 throughput = args.pathSegments[idx].throughput;
    if (SolveDirectLighting(args.sceneData, intersection, view_point, rng, directRadiance, wiW_Li, pdf_Li))
    {
        float bsdf_pdf = Pdf(material.type, intersection.surfaceNormal, -path.ray.direction, wiW_Li);
        float lambert_Li = glm::abs(glm::dot(intersection.surfaceNormal, wiW_Li));
        glm::vec3 matBsdf = f_diffuse(material.color); // TODO: Support diff materials?

        // Assemble direct lighting components
        glm::vec3 directLightResult = args.pathSegments[idx].throughput * directRadiance * lambert_Li / pdf_Li;
        thisBounceRadiance += directLightResult * PowerHeuristic(1, pdf_Li, 1, bsdf_pdf);
        args.pathSegments[idx].Lo += thisBounceRadiance;
    }
}

__global__ void skSpecular(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 view_point = path.ray.origin + intersection.t * path.ray.direction;
    glm::vec3 wiW_bsdf;
    float pdf_bsdf; // for spec materials, this should be 1.0
    glm::vec3 bsdf = Sample_f_specular(material.color, path.ray.direction, intersection.surfaceNormal, wiW_bsdf, pdf_bsdf);

    // Spec bounces don't need to do direct light calculation, since they only reflect light in one direction.

    args.pathSegments[idx].throughput *= material.color;
    args.pathSegments[idx].ray = SpawnRay(view_point, wiW_bsdf);
    args.pathSegments[idx].prevBounceSample.pdf = pdf_bsdf;
    args.pathSegments[idx].prevBounceSample.matType = MT_SPECULAR;
    args.pathSegments[idx].remainingBounces--;
}

__global__ void skEmissive(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    assert(material.type == MT_EMISSIVE);
    
    glm::vec3 totalRadiance(0.0f);
    glm::vec3 throughput = args.pathSegments[idx].throughput;
    if (args.depth == 0 || path.prevBounceSample.matType == MT_SPECULAR) // If this is the first bounce or we just came from specular, there is no "previous" data to go off
    {
        totalRadiance = material.color * material.emittance * throughput;
    }
    else
    {
        if (intersection.hitGeomIdx == -1)
        {
            // Error: this intersection should have geometry associated with it.
            args.pathSegments[idx].Lo = glm::vec3(0.0f);
            args.pathSegments[idx].remainingBounces = 0;
            return;
        }
        // TODO: Check that path.previous is not specular
        glm::mat4 geomTfm = args.sceneData.geoms[intersection.hitGeomIdx].transform; // TODO: Maybe just hold the tfm (and type) instead.
        thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);
        thrust::uniform_int_distribution<float> uH(-0.5f, 0.5f);
        glm::vec4 randPosLocal(uH(rng), uH(rng), 0.0f, 1.0f);
        glm::vec3 randPosWorld = glm::vec3(geomTfm * randPosLocal);

        float lightPdf = Pdf_Rect(geomTfm, path.ray.origin, randPosWorld, intersection.surfaceNormal);
        float bsdfPdf = path.prevBounceSample.pdf;
        totalRadiance += (material.color * material.emittance) * throughput * PowerHeuristic(1, bsdfPdf, 1, lightPdf);
    }
    volatile glm::vec3 loCopy = args.pathSegments[idx].Lo;
    args.pathSegments[idx].Lo += totalRadiance;
    args.pathSegments[idx].remainingBounces = 0; // Mark it for culling later
}

__global__ void skRefractive(ShadeKernelArgs args)
{
    return; // TODO
}

#if ONLY_BSDF_SAMPLING
__global__ void skDiffuseSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);

    glm::vec3 wi = calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng);
    glm::vec3 bsdf = f_diffuse(material.color);
    glm::vec3 lightTransportResult = bsdf * PI; // Normally (bsdf*lambert)/pdf but this is simplified

    args.pathSegments[idx].throughput *= lightTransportResult;
    args.pathSegments[idx].ray = SpawnRay(path.ray.origin + intersection.t * path.ray.direction, wi);
    args.pathSegments[idx].remainingBounces--;
}

__global__ void skSpecularSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 wiW = glm::reflect(path.ray.direction, intersection.surfaceNormal);
    args.pathSegments[idx].throughput *= material.color;
    args.pathSegments[idx].ray = SpawnRay(path.ray.origin + intersection.t * path.ray.direction, wiW);
    args.pathSegments[idx].remainingBounces--;
}

#endif

#if DIRECT_SAMPLING
__global__ void skDiffuseDirect(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];
    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);

    HANDLE_MISS(idx, intersection, pathSegments);

    Light* lights = args.sceneData.lights;
    int numLights = args.sceneData.lights_size;
    thrust::uniform_int_distribution<int> iu0N(0, numLights - 1);
    glm::vec3 wiW;
    float pdf;
    glm::vec3 view_point = path.ray.origin + (intersection.t * path.ray.direction);
    glm::vec3 totalDirectLight(0.0f);
    glm::vec3 bsdf = f_diffuse(material.color);
    const int NUM_SAMPLES = 4;
    for (int s = 0; s != NUM_SAMPLES; ++s)
    {
        glm::vec3 radiance;
        if (!SolveDirectLighting(args.sceneData, intersection, view_point, rng, radiance, wiW, pdf))
            continue;

        float cosTheta = glm::dot(wiW, intersection.surfaceNormal);
        if (cosTheta < FLT_EPSILON)
            continue;

        totalDirectLight += radiance * cosTheta / (NUM_SAMPLES * pdf);
    }
    totalDirectLight *= numLights;

    args.pathSegments[idx].throughput *= bsdf;
    glm::vec3 throughput = args.pathSegments[idx].throughput;

    args.pathSegments[idx].Lo += throughput * totalDirectLight;
    args.pathSegments[idx].remainingBounces = 0;
}
#endif

#if DIRECT_SAMPLING || ONLY_BSDF_SAMPLING
__global__ void skEmissiveSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 throughput = args.pathSegments[idx].throughput;
    args.pathSegments[idx].Lo += (material.color * material.emittance) * throughput;
    args.pathSegments[idx].remainingBounces = 0; // Mark it for culling later
}
#endif

// By convention: MUST match the order of the MaterialType struct

#if MIS_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuse,
    skSpecular,
    skEmissive,
    skRefractive
};
#elif DIRECT_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuseDirect,
    skSpecular,
    skEmissiveSimple,
    skRefractive
};
#else ONLY_BSDF_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuseSimple,
    skSpecular,
    skEmissiveSimple,
    skRefractive
};
#endif

__host__ ShadeKernel getShadingKernelForMaterial(MaterialType mt)
{
    assert(mt < MT_COUNT);
    return sKernels[mt];
}