#include "hip/hip_runtime.h"
#include "bsdf.h"

#include "light.h"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

__device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

__host__ __device__ inline bool isnanVec3(const glm::vec3& v) {
    return isnan(v.x) || isnan(v.y) || isnan(v.z);
}

inline __device__ void coordinateSystem(const glm::vec3& v1, glm::vec3& v2, glm::vec3& v3)
{
    if (glm::abs(v1.x) > glm::abs(v1.y))
        v2 = glm::vec3(-v1.z, 0, v1.x) / glm::sqrt(v1.x * v1.x + v1.z * v1.z);
    else
        v2 = glm::vec3(0, v1.z, -v1.y) / glm::sqrt(v1.y * v1.y + v1.z * v1.z);
    v3 = glm::cross(v1, v2);
}

inline __device__ glm::mat3 LocalToWorld(glm::vec3 nor)
{
    glm::vec3 tan, bit;
    coordinateSystem(nor, tan, bit);
    return glm::mat3(tan, bit, nor);
}


inline __device__ glm::mat3 WorldToLocal(glm::vec3 nor) {
    return glm::transpose(LocalToWorld(nor));
}

inline __device__ Ray SpawnRay(const glm::vec3& pos, const glm::vec3& wi)
{
    Ray r;
    r.origin = pos + wi * 0.001f;
    r.direction = wi;
    return r;
}

inline __device__ glm::vec4 TextureSample(hipTextureObject_t texObj, const glm::vec2& uv)
{
    float4 color = tex2D<float4>(texObj, uv.x, uv.y);
    return glm::vec4(color.x, color.y, color.z, color.w);
}

inline __device__ glm::vec4 TryTextureSample(hipTextureObject_t* texObjArr, int textureId, const glm::vec2& uv, const glm::vec4& fallbackValue)
{
    if (textureId == -1)
        return fallbackValue;

    return TextureSample(texObjArr[textureId], uv);
}

inline __device__ glm::vec3 TryTextureSample(hipTextureObject_t* texObjArr, int textureId, const glm::vec2& uv, const glm::vec3& fallbackValue)
{
    return glm::vec3(TryTextureSample(texObjArr, textureId, uv, glm::vec4(fallbackValue, 0.0f)));
}

////////////////////////
// PBR Utils (Mostly from 561)

__device__ float pow5(float x) {
    float x2 = x * x;
    return x2 * x2 * x;
}

// Schlick's Fresnel approximation
//__device__ glm::vec3 fresnelSchlick(float cosTheta, const glm::vec3& F0) {
//    return F0 + (glm::vec3(1.0f) - F0) * pow5(glm::clamp(1.0f - cosTheta, 0.0f, 1.0f));
//}

__device__ glm::vec3 fresnelSchlickRoughness(float cosTheta, glm::vec3 F0, float roughness)
{
    return F0 + (glm::max(glm::vec3(1.0f - roughness), F0) - F0) * pow5(glm::clamp(1.0f - cosTheta, 0.0f, 1.0f));
}

// GGX/Trowbridge-Reitz Normal Distribution Function
__device__ float distributionGGX(const glm::vec3& norW, const glm::vec3& whW, float roughness) {
    float a = roughness * roughness;
    float a2 = a * a;
    float NdotH = glm::max(glm::dot(norW, whW), 0.0f);
    float NdotH2 = NdotH * NdotH;

    float denom = (NdotH2 * (a2 - 1.0f) + 1.0f);
    denom = PI * denom * denom;
    denom = glm::max(denom, 0.01f);

    return a2 / denom;
}

__device__ glm::vec3 sampleGGX(const glm::vec3& norW, float roughness, thrust::default_random_engine& rng) {
    
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    float u1 = u01(rng);
    float u2 = u01(rng);
    
    float a = roughness * roughness;
    float a2 = a * a;

    float phi = 2.0f * PI * u1;

    // Spherical coords
    float tanTheta2 = a2 * u2 / (1.0f - u2 + FLT_EPSILON);
    float cosTheta = 1.0f / sqrt(1.0f + tanTheta2);
    float sinTheta = sqrt(glm::max(0.0f, 1.0f - cosTheta * cosTheta));

    assert(!isnan(cosTheta) && !isnan(sinTheta));

    // Cartesian
    glm::vec3 H;
    H.x = cos(phi) * sinTheta;
    H.y = sin(phi) * sinTheta;
    H.z = cosTheta;

    // Build TBN frame
    glm::vec3 up = abs(norW.z) < 0.999f ? glm::vec3(0.0f, 0.0f, 1.0f) : glm::vec3(1.0f, 0.0f, 0.0f);
    glm::vec3 tangent = glm::normalize(glm::cross(up, norW));
    glm::vec3 bitangent = glm::cross(norW, tangent);

    glm::vec3 result = glm::normalize(tangent * H.x + bitangent * H.y + norW * H.z);

    assert(!isnanVec3(result));
    return result;
}

// Smith's Geometry function with GGX (Schlick-GGX)
__device__ float geometrySchlickGGX(float NdotV, float roughness) {
    float r = (roughness + 1.0f);
    float k = (r * r) / 8.0f;

    return NdotV / (NdotV * (1.0f - k) + k);
}

__device__ float geometrySmith(const glm::vec3& norW, const glm::vec3& woW, const glm::vec3& wiW, float roughness) {
    float NdotWo = glm::max(glm::dot(norW, woW), 0.1f);
    float NdotWi = glm::max(glm::dot(norW, wiW), 0.1f);
    float ggx2 = geometrySchlickGGX(NdotWo, roughness);
    float ggx1 = geometrySchlickGGX(NdotWi, roughness);

    return ggx1 * ggx2;
}
////////////////////////
// PDF functions

__device__ float squareToHemisphereCosinePDF(const glm::vec3& sampleL)
{
    return sampleL.z * INV_PI;
}

__device__ float pdfGGX(const glm::vec3& norW, const glm::vec3& whW, const glm::vec3& woW, float roughness) {
    float HdotN  = glm::max(glm::dot(whW, norW), 0.0f);
    float HdotWo = glm::max(glm::dot(whW, woW), 0.01f);
    
    float D = distributionGGX(norW, whW, roughness);


    return (D * HdotN) / (4.0f * HdotWo);
}

__device__ float pdfCookTorrance(const glm::vec3& norW, const glm::vec3& whW, const glm::vec3& woW, const glm::vec3& wiW, float roughness, float metallic) {
    // Mix between diffuse and specular PDF based on Fresnel
    float VdotH = glm::max(glm::dot(woW, whW), 0.01f);
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), glm::vec3(1.0f), metallic);
    glm::vec3 F = fresnelSchlickRoughness(VdotH, F0, roughness);
    float specularWeight = (F.x + F.y + F.z) / 3.0f;

    float pdfSpec = pdfGGX(norW, whW, woW, roughness);
    float pdfDiff = glm::max(glm::dot(norW, wiW), 0.0f) * INV_PI;

    assert(!isnan(pdfSpec) && !isnan(pdfDiff) && !isnan(specularWeight));

    return glm::mix(pdfDiff, pdfSpec, specularWeight);
}

__device__ float Pdf(MaterialType matType, glm::vec3 norW, glm::vec3 woW, glm::vec3 wiW)
{
    glm::vec3 nor = norW;
    glm::mat3 world2Local = WorldToLocal(nor);
    glm::vec3 wo = world2Local * woW;
    glm::vec3 wi = world2Local * wiW;

    switch (matType)
    {
    case MT_DIFFUSE:
        return squareToHemisphereCosinePDF(wi);
    case MT_MICROFACET_PBR:
    {
        //glm::vec3 whW = glm::normalize(woW + wiW);
        //return pdfCookTorrance(norW, whW, woW, wiW, 0.1f, 0.1f);
    }
    case MT_SPECULAR:
        return 0.0f; // Spec goes directly to the light, and refractive goes into the material.
    }

    return 0.0f; // Unhandled material.
}

////////////////////////
// f / Samplef functions

inline __device__ glm::vec3 f_diffuse(const glm::vec3& albedo)
{
    return albedo * INV_PI;
}

inline __device__ glm::vec3 f_spec(const glm::vec3& albedo, const glm::vec3& wiW, const glm::vec3& norW)
{
    float absCosTheta = glm::abs(glm::dot(wiW, norW));
    if (absCosTheta < FLT_EPSILON)
        return glm::vec3(0.0f);

    return albedo / absCosTheta;
}

inline __device__ glm::vec3 f_cookTorrance(const glm::vec3& albedo, const glm::vec3& norW, const glm::vec3& woW, const glm::vec3& wiW, float roughness, float metallic)
{
    glm::vec3 whW = glm::normalize(wiW + woW);

    float NdotWo = glm::max(glm::dot(norW, woW), 0.0f);
    float NdotWi = glm::max(glm::dot(norW, wiW), 0.0f);

    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), albedo, metallic);

    // Cook-Torrance specular
    float D = distributionGGX(norW, whW, roughness);
    float G = geometrySmith(norW, woW, wiW, roughness);
    glm::vec3 F = fresnelSchlickRoughness((glm::dot(whW, woW)), F0, roughness);

    glm::vec3 numerator = D * G * F;
    float denominator = 4.0f * NdotWi * NdotWo + 0.001f; // Add epsilon to prevent division by zero
    glm::vec3 specular = numerator / denominator;

    // Lambertian component
    // For metals, diffuse is 0 
    glm::vec3 kS = F; // Specular part
    glm::vec3 kD = glm::vec3(1.0f) - kS; // Diffuse part
    kD *= (1.0f - metallic); // Metals have no diffuse

    glm::vec3 diffuse = kD * albedo / PI;

    return (diffuse + specular) * NdotWo;
}

inline __device__ glm::vec3 Sample_f_diffuse(const glm::vec3& albedo, const glm::vec3& norW, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf)
{
    out_wiW = calculateRandomDirectionInHemisphere(norW, rng);
    out_pdf = glm::abs(glm::dot(out_wiW, norW)) * INV_PI;
    return f_diffuse(albedo);
}

inline __device__ glm::vec3 Sample_f_specular(const glm::vec3& albedo, const glm::vec3& woW, const glm::vec3& norW, glm::vec3& out_wiW, float& out_pdf)
{
    out_wiW = glm::reflect(woW, norW);
    out_pdf = 1.0f;
    return f_spec(albedo, out_wiW, norW);
}

__device__ glm::vec3 Sample_f_cookTorrance(const glm::vec3& albedo, const glm::vec3& woW, const glm::vec3& norW, float metallic, float roughness, thrust::default_random_engine& rng, glm::vec3& out_wiW, float& out_pdf)
{
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), albedo, metallic);

    float NdotWo = glm::max(glm::dot(woW, norW), 0.0f);
    glm::vec3 F = fresnelSchlickRoughness(NdotWo, F0, roughness);
    float specWeight = (F.x + F.y + F.z) / 3.0f; // Avg of each component, used to choose between spec and diffuse

    glm::vec3 wiW(0.0f);
    glm::vec3 whW(0.0f); // resulting view vector and half vector

    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    // Choose spec vs diffuse based on random number and spec weight
    float u1 = u01(rng);
    if (u1 < specWeight)
    {
        whW = sampleGGX(norW, roughness, rng);
        wiW = glm::reflect(-woW, whW);

        assert(!isnanVec3(wiW) && !isnanVec3(whW));

        // hemisphere check
        if (glm::dot(wiW, norW) <= 0.0f)
        {
            out_pdf = 1.0f;
            return glm::vec3(1.0f);
        }
    }
    else
    {
        // Just diffuse sampling
        wiW = calculateRandomDirectionInHemisphere(norW, rng);
        whW = glm::normalize(woW + wiW);
    }


    assert(!isnanVec3(wiW) && !isnanVec3(whW));

    float pdf = pdfCookTorrance(norW, whW, woW, wiW, roughness, metallic);
    if (pdf < FLT_EPSILON)
    {
        out_pdf = 1.0f;
        return glm::vec3(1.0f);
    }

    out_wiW = wiW;
    out_pdf = pdf;
    return f_cookTorrance(albedo, norW, woW, wiW, roughness, metallic);
}

__device__ bool SolveDirectLighting(const SceneData& sd, hipTextureObject_t envMapObj, const glm::vec3& norW, glm::vec3 view_point, thrust::default_random_engine& rng, glm::vec3& out_radiance, glm::vec3& out_wiW, float& out_pdf)
{
    envMapObj = 0;
    int numLights = sd.lights_size;
    //if (numLights <= 0)
    //    return false;

    int randomLightIndex = 0;

    if (numLights > 0)
    {
        thrust::uniform_int_distribution<int> iu0N(0, envMapObj != 0 ? numLights : numLights-1);
        randomLightIndex = iu0N(rng);
    }

    float pdf_Li;
    float distToLight;
    glm::vec3 wiW_Li;
    int ignoreGeomId = -1;
    glm::vec3 liResult;

    if (randomLightIndex == numLights) // This is out of bounds for the array, this represents the environment map.
    {
        // Like diffuse bsdf, just choose a random direction in our local hemisphere.
        wiW_Li = calculateRandomDirectionInHemisphere(norW, rng);
        pdf_Li = glm::abs(glm::dot(wiW_Li, norW)) * INV_PI;

        liResult = sampleEnvironmentMap(envMapObj, wiW_Li);
    }
    else
    {
        const Light chosenLight = sd.lights[randomLightIndex];
        ignoreGeomId = chosenLight.geomId;

        liResult = Sample_Li(view_point, norW, chosenLight, numLights, rng, wiW_Li, pdf_Li, distToLight);
        if (pdf_Li < FLT_EPSILON)
            return false;
    }

    // Test occlusion
    PathSegment shadowPath;
    shadowPath.ray = SpawnRay(view_point, wiW_Li);
    ShadeableIntersection shadowTestResult;
    sceneIntersect(shadowPath, sd, shadowTestResult, nullptr, ignoreGeomId);
    
    if (shadowTestResult.t >= 0.0f && shadowTestResult.t < (distToLight - FLT_EPSILON))
        return false;

    out_radiance = liResult;
    out_wiW = wiW_Li;
    out_pdf = pdf_Li;

    return true;
}

/////////

#if STREAM_COMPACTION
#define HANDLE_MISS(idx, intersection, pathSegments) \
        assert((intersection).t > FLT_EPSILON);
#else
#define HANDLE_MISS(idx, intersection, pathSegments)              \
        if ((intersection).t <= 0.0f) {                               \
            (pathSegments)[(idx)].color = glm::vec3(0.0f);            \
            (pathSegments)[(idx)].remainingBounces = 0;               \
            return;                                                   \
        }
#endif


__global__ void skDiffuse(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);
    glm::vec3 view_point = path.ray.origin + intersection.t * path.ray.direction;
    glm::vec3 thisBounceRadiance(0.0f); // Comes from direct lighting only
    
    const glm::vec3 ERROR_COLOR(1.0f, 0.4118f, 0.7059f);
    glm::vec3 wiW_bsdf;
    float pdf_bsdf;
    glm::vec3 bsdf;

    glm::vec3 albedo;
    if (material.diffuseTexId != -1)
    {
        hipTextureObject_t texObj = args.textures[material.diffuseTexId];
        if (!texObj)
            albedo = ERROR_COLOR;
        else
            albedo = glm::vec3(TextureSample(texObj, intersection.uv));
    }
    else
    {
        albedo = material.color;
    }
    
    // BSDF Sampling
    glm::vec3 norW = intersection.surfaceNormal;
    bsdf = Sample_f_diffuse(albedo, norW, rng, wiW_bsdf, pdf_bsdf);
    if (pdf_bsdf < FLT_EPSILON)
    {
        // Something went wrong, terminate
        args.pathSegments[idx].remainingBounces = 0;
        return;
    }

    glm::vec3 directRadiance;
    glm::vec3 wiW_Li;
    float pdf_Li;

    float lambert = glm::abs(glm::dot(norW, wiW_bsdf));
    args.pathSegments[idx].throughput *= (bsdf/pdf_bsdf) * lambert;
    args.pathSegments[idx].prevBounceSample.pdf = pdf_bsdf;
    args.pathSegments[idx].prevBounceSample.matType = MT_DIFFUSE;
    args.pathSegments[idx].ray = SpawnRay(view_point, wiW_bsdf);
    args.pathSegments[idx].remainingBounces--;

    // Direct Light Sampling
    // Key difference using MIS: Accumulate direct lighting radiance here.
    glm::vec3 throughput = args.pathSegments[idx].throughput;
    hipTextureObject_t envMapObj = args.envMaps ? args.envMaps[0] : 0;
    if (SolveDirectLighting(args.sceneData, envMapObj, norW, view_point, rng, directRadiance, wiW_Li, pdf_Li))
    {
        float bsdf_pdf = Pdf(material.type, norW, -path.ray.direction, wiW_Li);
        float lambert_Li = glm::abs(glm::dot(norW, wiW_Li));

        // Assemble direct lighting components
        glm::vec3 directLightResult = args.pathSegments[idx].throughput * directRadiance * lambert_Li / pdf_Li;
        thisBounceRadiance += directLightResult * PowerHeuristic(1, pdf_Li, 1, bsdf_pdf);
        args.pathSegments[idx].Lo += thisBounceRadiance;
    }
}

__global__ void skSpecular(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 view_point = path.ray.origin + intersection.t * path.ray.direction;
    glm::vec3 wiW_bsdf;
    float pdf_bsdf; // for spec materials, this should be 1.0
    glm::vec3 bsdf = Sample_f_specular(material.color, path.ray.direction, intersection.surfaceNormal, wiW_bsdf, pdf_bsdf);

    // Spec bounces don't need to do direct light calculation, since they only reflect light in one direction.
    float lambert = glm::abs(glm::dot(intersection.surfaceNormal, wiW_bsdf));
    args.pathSegments[idx].throughput *= (bsdf * lambert) / pdf_bsdf;
    args.pathSegments[idx].ray = SpawnRay(view_point, wiW_bsdf);
    args.pathSegments[idx].prevBounceSample.pdf = pdf_bsdf;
    args.pathSegments[idx].prevBounceSample.matType = MT_SPECULAR;
    args.pathSegments[idx].remainingBounces--;
}

__global__ void skEmissive(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    assert(material.type == MT_EMISSIVE);
    
    glm::vec3 totalRadiance(0.0f);
    glm::vec3 throughput = args.pathSegments[idx].throughput;
    if (args.depth == 0 || path.prevBounceSample.matType == MT_SPECULAR) // If this is the first bounce or we just came from specular, there is no "previous" data to go off
    {
        totalRadiance = material.color * material.emittance * throughput;
    }
    else
    {
        if (intersection.hitGeomIdx == -1)
        {
            // Error: this intersection should have geometry associated with it.
            args.pathSegments[idx].Lo = glm::vec3(0.0f);
            args.pathSegments[idx].remainingBounces = 0;
            return;
        }
        // TODO: Check that path.previous is not specular
        glm::mat4 geomTfm = args.sceneData.geoms[intersection.hitGeomIdx].transform; // TODO: Maybe just hold the tfm (and type) instead.
        thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);
        thrust::uniform_int_distribution<float> uH(-0.5f, 0.5f);
        glm::vec4 randPosLocal(uH(rng), uH(rng), 0.0f, 1.0f);
        glm::vec3 randPosWorld = glm::vec3(geomTfm * randPosLocal);

        float lightPdf = Pdf_Rect(geomTfm, path.ray.origin, randPosWorld, intersection.surfaceNormal);
        float bsdfPdf = path.prevBounceSample.pdf;
        totalRadiance += (material.color * material.emittance) * throughput * PowerHeuristic(1, bsdfPdf, 1, lightPdf);
    }

    args.pathSegments[idx].Lo += totalRadiance;
    args.pathSegments[idx].remainingBounces = 0; // Mark it for culling later
}

__global__ void skMicrofacetPBR(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);
    glm::vec3 view_point = path.ray.origin + intersection.t * path.ray.direction;
    glm::vec3 thisBounceRadiance(0.0f); // Comes from direct lighting only

    const glm::vec4 metallicRoughFallback(material.metallic, material.roughness, 1.0f, 1.0f);

    glm::vec3 woW = -path.ray.direction;

    glm::vec3 albedo = TryTextureSample(args.textures, material.diffuseTexId, intersection.uv, material.color);
    glm::vec4 metallicRough = TryTextureSample(args.textures, material.metallicRoughTexId, intersection.uv, metallicRoughFallback);

    glm::vec3 norW = intersection.surfaceNormal;
    
    if (material.normalTexId != -1)
    {
        glm::vec3 sampledNormal = glm::vec3(TextureSample(args.textures[material.normalTexId], intersection.uv));
        glm::vec3 norT = (2.0f * sampledNormal) - glm::vec3(1.0f);

        glm::vec3 tangent = intersection.tangent;
        glm::vec3 bitangent = glm::normalize(glm::cross(norW, tangent));

        // Ensure right-handed coordinate system
        if (glm::dot(glm::cross(norW, tangent), bitangent) < 0.0f) {
            tangent = -tangent;
        }

        glm::mat3 TBN(tangent, bitangent, norW);
        norW = glm::normalize(TBN * norT);
    }
    
    // This assumes the metallic and roughness always go in the same channels as they should
    // Someone should let the people on sketchfab know....
    float roughness = glm::clamp(metallicRough.g, 0.01f, 1.0f);
    float metallic = glm::clamp(metallicRough.b, 0.01f, 1.0f);

    glm::vec3 wiW;
    float pdf;
    glm::vec3 bsdf = Sample_f_cookTorrance(albedo, woW, norW, metallic, roughness, rng, wiW, pdf);
    if (pdf < FLT_EPSILON)
    {
        // Something went wrong, terminate
        args.pathSegments[idx].remainingBounces = 0;
        return;
    }

    bsdf = glm::clamp(bsdf, glm::vec3(0.0f), glm::vec3(1.0f));

    //assert(bsdf.x > 0 && bsdf.y > 0 && bsdf.z > 0);
    assert(!isnanVec3(bsdf));

    glm::vec3 directRadiance;
    glm::vec3 wiW_Li;
    float pdf_Li;

    float lambert = glm::abs(glm::dot(norW, wiW));
    args.pathSegments[idx].throughput *= (bsdf / pdf);
    args.pathSegments[idx].prevBounceSample.pdf = pdf;
    args.pathSegments[idx].prevBounceSample.matType = MT_MICROFACET_PBR;
    args.pathSegments[idx].ray = SpawnRay(view_point, wiW);
    args.pathSegments[idx].remainingBounces--;

    // Direct Light Sampling
    // Key difference using MIS: Accumulate direct lighting radiance here.
    hipTextureObject_t envMapObj = args.envMaps ? args.envMaps[0] : 0;
    glm::vec3 throughput = args.pathSegments[idx].throughput;
    if (SolveDirectLighting(args.sceneData, envMapObj, norW, view_point, rng, directRadiance, wiW_Li, pdf_Li))
    {
        glm::vec3& whW = glm::normalize(woW + wiW_Li);
        float bsdf_pdf = pdfCookTorrance(norW, whW, woW, wiW_Li, roughness, metallic);
        float lambert_Li = glm::abs(glm::dot(norW, wiW_Li));

        // Assemble direct lighting components
        glm::vec3 directLightResult = args.pathSegments[idx].throughput * directRadiance * lambert_Li / pdf_Li;
        thisBounceRadiance += directLightResult * PowerHeuristic(1, pdf_Li, 1, bsdf_pdf);
        args.pathSegments[idx].Lo += thisBounceRadiance;
    }
}

#if ONLY_BSDF_SAMPLING
__global__ void skDiffuseSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);

    glm::vec3 wi = calculateRandomDirectionInHemisphere(intersection.surfaceNormal, rng);
    glm::vec3 bsdf = f_diffuse(material.color);
    glm::vec3 lightTransportResult = bsdf * PI; // Normally (bsdf*lambert)/pdf but this is simplified

    args.pathSegments[idx].throughput *= lightTransportResult;
    args.pathSegments[idx].ray = SpawnRay(path.ray.origin + intersection.t * path.ray.direction, wi);
    args.pathSegments[idx].remainingBounces--;
}

__global__ void skSpecularSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 wiW = glm::reflect(path.ray.direction, intersection.surfaceNormal);
    args.pathSegments[idx].throughput *= material.color;
    args.pathSegments[idx].ray = SpawnRay(path.ray.origin + intersection.t * path.ray.direction, wiW);
    args.pathSegments[idx].remainingBounces--;
}

#endif

#if DIRECT_SAMPLING
__global__ void skDiffuseDirect(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];
    thrust::default_random_engine rng = makeSeededRandomEngine(args.iter, idx, path.remainingBounces);

    HANDLE_MISS(idx, intersection, pathSegments);

    Light* lights = args.sceneData.lights;
    int numLights = args.sceneData.lights_size;
    thrust::uniform_int_distribution<int> iu0N(0, numLights - 1);
    glm::vec3 wiW;
    float pdf;
    glm::vec3 view_point = path.ray.origin + (intersection.t * path.ray.direction);
    glm::vec3 totalDirectLight(0.0f);
    glm::vec3 bsdf = f_diffuse(material.color);
    const int NUM_SAMPLES = 4;
    for (int s = 0; s != NUM_SAMPLES; ++s)
    {
        glm::vec3 radiance;
        if (!SolveDirectLighting(args.sceneData, intersection, view_point, rng, radiance, wiW, pdf))
            continue;

        float cosTheta = glm::dot(wiW, intersection.surfaceNormal);
        if (cosTheta < FLT_EPSILON)
            continue;

        totalDirectLight += radiance * cosTheta / (NUM_SAMPLES * pdf);
    }
    totalDirectLight *= numLights;

    args.pathSegments[idx].throughput *= bsdf;
    glm::vec3 throughput = args.pathSegments[idx].throughput;

    args.pathSegments[idx].Lo += throughput * totalDirectLight;
    args.pathSegments[idx].remainingBounces = 0;
}
#endif

#if DIRECT_SAMPLING || ONLY_BSDF_SAMPLING
__global__ void skEmissiveSimple(ShadeKernelArgs args)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= args.num_paths)
        return;

    const PathSegment path = args.pathSegments[idx];
    const ShadeableIntersection intersection = args.shadeableIntersections[idx];
    const Material material = args.materials[GetMaterialIDFromSortKey(intersection.matSortKey)];

    HANDLE_MISS(idx, intersection, pathSegments);

    glm::vec3 throughput = args.pathSegments[idx].throughput;
    args.pathSegments[idx].Lo += (material.color * material.emittance) * throughput;
    args.pathSegments[idx].remainingBounces = 0; // Mark it for culling later
}
#endif

// By convention: MUST match the order of the MaterialType struct

#if MIS_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuse,
    skSpecular,
    skEmissive,
    skMicrofacetPBR
};
#elif DIRECT_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuseDirect,
    skSpecular,
    skEmissiveSimple,
    skMicrofacetPBR
};
#else ONLY_BSDF_SAMPLING
static ShadeKernel sKernels[] =
{
    skDiffuseSimple,
    skSpecular,
    skEmissiveSimple,
    skMicrofacetPBR
};
#endif

__host__ ShadeKernel getShadingKernelForMaterial(MaterialType mt)
{
    assert(mt < MT_COUNT);
    return sKernels[mt];
}